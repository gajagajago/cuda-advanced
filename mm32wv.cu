#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <cassert>
#include <hipblas.h>

#define EPS 1e-3 
#define CHECK_CUDA(e) \
  if ((e) != hipSuccess) { \
    printf("[%s:%d CudaError]: %s\n", \
        __FILE__, __LINE__, hipGetErrorString(e)); \
    exit(EXIT_FAILURE);                          \
  } 
#define CHECK_CUBLAS(e)                                  \
  if ((e) != HIPBLAS_STATUS_SUCCESS) {                    \
    printf("[%s:%d CublasError]\n", __FILE__, __LINE__); \
    exit(EXIT_FAILURE);                                  \
  }

#define MAX(a, b) (((a) < (b)) ? (b) : (a))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

#define WARMUP

#define V (4)

float *A, *B, *C, *C_ans;
float *A_cuda, *B_cuda, *C_cuda, *C_cublas;

constexpr int M = 4096;
constexpr int K = 4096;
constexpr int N = 4096;

//////////////////////////////////////////// DO NOT MODIFY ////////////////////////////////////////////
constexpr int BLOCK_M = 16;
constexpr int BLOCK_N = 16;

constexpr int BLOCK_TILE_M = 128; // d_bm
constexpr int BLOCK_TILE_N = 128; // d_bn
constexpr int BLOCK_TILE_K = 16;

/* SMEM load */
constexpr int A_N = MAX(MIN(BLOCK_TILE_K / V, BLOCK_N), 1); // A tile row is loaded by BLOCK_N threads, and can take multiple iterations
constexpr int A_M = (BLOCK_M * BLOCK_N) / A_N;    // Number of A tile rows loaded by a thread block in a single iteration
constexpr int B_N = MAX(MIN(BLOCK_TILE_N / V, BLOCK_N), 1); // B tile row is loaded by BLOCK_N threads, and can take multiple iterations
constexpr int B_M = (BLOCK_M * BLOCK_N) / B_N;    // Number of A tile rows loaded by a thread block in a single iteration

/* Warps for a block tile */
constexpr int WARP_TILE_M = 64; // d_wm
constexpr int WARP_TILE_N = 32; // d_wn

static_assert(BLOCK_TILE_M % WARP_TILE_M == 0);
static_assert(BLOCK_TILE_N % WARP_TILE_N == 0);

constexpr int NUM_WARPS_PER_BLOCK_TILE_M = BLOCK_TILE_M / WARP_TILE_M;  // NUM_WARPS_Y
constexpr int NUM_WARPS_PER_BLOCK_TILE_N = BLOCK_TILE_N / WARP_TILE_N;  // NUM_WARPS_X

constexpr int NUM_THREADS_PER_WARP_Y = 8; // m_t
constexpr int NUM_THREADS_PER_WARP_X = 4; // n_t

static_assert(NUM_THREADS_PER_WARP_Y * NUM_THREADS_PER_WARP_X == 32);
////////////////////////////////////////////////////////////////////////////////////////////////////////

/* Threads for a warp tile */
constexpr int THREAD_TILE_M = 8;  // d_tm
constexpr int THREAD_TILE_N = 8;  // d_tn

static_assert(WARP_TILE_M % (THREAD_TILE_M * NUM_THREADS_PER_WARP_Y) == 0); // (d_wm / m_t) % d_tm == 0
static_assert(WARP_TILE_N % (THREAD_TILE_N * NUM_THREADS_PER_WARP_X) == 0); // (d_wn / n_t) % d_tn == 0
static_assert(THREAD_TILE_N % V == 0);

__global__ void mm32wv(float4 *A, float4 *B, float4 *C, const int M, const int K, const int N)
{
  if (blockIdx.x * BLOCK_TILE_N >= N || blockIdx.y * BLOCK_TILE_M >= M) return;

  const int K_V = K / V;
  const int N_V = N / V;
  const int BLOCK_TILE_K_V = BLOCK_TILE_K / V;
  const int BLOCK_TILE_N_V = BLOCK_TILE_N / V;
  const int WARP_TILE_N_V = WARP_TILE_N / V;
  const int THREAD_TILE_N_V = THREAD_TILE_N / V;

  const float4 ZERO = { 0.f };

  __shared__ float4 A_shared[BLOCK_TILE_M][BLOCK_TILE_K_V]; // TODO: Use transposed to group four A elements in a column as vector
  __shared__ float4 B_shared[BLOCK_TILE_K][BLOCK_TILE_N_V];

  // Each thread computes (d_wm / m_t / d_tm) * (d_wn / n_t / d_tn) number of tiles,
  // each with size d_tm x d_tn
  constexpr int NUM_THREAD_TILES_PER_WARP_TILE_M = WARP_TILE_M / NUM_THREADS_PER_WARP_Y / THREAD_TILE_M; // Thread가 속한 warp의 tile 중, 내 thread tile로 계산하는 개수
  constexpr int NUM_THREAD_TILES_PER_WARP_TILE_N = WARP_TILE_N / NUM_THREADS_PER_WARP_X / THREAD_TILE_N; // Thread가 속한 warp의 tile 중, 내 thread tile로 계산하는 개수
  static_assert(WARP_TILE_M % (NUM_THREADS_PER_WARP_Y * THREAD_TILE_M) == 0);
  static_assert(WARP_TILE_N % (NUM_THREADS_PER_WARP_X * THREAD_TILE_N) == 0);

  float4 a_reg[NUM_THREAD_TILES_PER_WARP_TILE_M][THREAD_TILE_M] = { ZERO };
  float4 b_reg[NUM_THREAD_TILES_PER_WARP_TILE_N][V][THREAD_TILE_N_V] = { ZERO };
  float4 c_reg[NUM_THREAD_TILES_PER_WARP_TILE_M][NUM_THREAD_TILES_PER_WARP_TILE_N][THREAD_TILE_M][THREAD_TILE_N_V] = { ZERO };

  /* Loading */
  const int ay = (blockDim.x * threadIdx.y + threadIdx.x) / A_N;
  const int ax = (blockDim.x * threadIdx.y + threadIdx.x) % A_N;
  const int by = (blockDim.x * threadIdx.y + threadIdx.x) / B_N;
  const int bx = (blockDim.x * threadIdx.y + threadIdx.x) % B_N;

  /* Computing */
  const int THREAD_IDX = threadIdx.y * blockDim.x + threadIdx.x;  // tid in block
  const int WARP_IDX = THREAD_IDX / 32; // warp id in block
  const int WARP_Y = WARP_IDX / NUM_WARPS_PER_BLOCK_TILE_N; // warp y id in computing block tile
  const int WARP_X = WARP_IDX % NUM_WARPS_PER_BLOCK_TILE_N; // warp x id in computing block tile
  const int THREAD_Y_IN_WARP = (THREAD_IDX % 32) / NUM_THREADS_PER_WARP_X; // thread y id in warp tile
  const int THREAD_X_IN_WARP = (THREAD_IDX % 32) % NUM_THREADS_PER_WARP_X; // thread x id in warp tile

  for (int tk = 0; tk < K; tk += BLOCK_TILE_K) {
    // load A
    for (int ii = 0; ii < BLOCK_TILE_M; ii += A_M) {
      int li = ii + ay; // which row in shared A
      int Ai = BLOCK_TILE_M * blockIdx.y + li;  // which row in A
      for (int kk = 0; kk < BLOCK_TILE_K_V; kk += A_N) {  // load A row iteratively
        int lk = kk + ax; // which col in shared A
        int Ak = (tk / V)  + lk;
        A_shared[li][lk] = (Ai < M && Ak < K_V) ? A[Ai * K_V + Ak] : ZERO;
      }
    }

    // load B
    for (int kk = 0; kk < BLOCK_TILE_K; kk += B_M) {
      int lk = kk + by; // which row in shared B
      int Bk = tk + lk;  // which row in B
      for (int jj = 0; jj < BLOCK_TILE_N_V; jj += B_N) {  // load B row iteratively
        int lj = jj + bx; // which col in shared B
        int Bj = blockIdx.x * (BLOCK_TILE_N_V) + lj;  // which col in B
        B_shared[lk][lj] = (Bk < K && Bj < N_V) ? B[Bk * N_V + Bj] : ZERO;
      }
    }
    // sync after load
    __syncthreads();

    // validate A
    // if (threadIdx.x == 0 && threadIdx.y == 0) {
    //   int Ai = BLOCK_TILE_M * blockIdx.y;
    //   for (int ii = 0; ii < BLOCK_TILE_M; ++ii) {
    //     int Ak = tk / V;
    //     for (int kk = 0; kk < BLOCK_TILE_K_V; ++kk) {
    //       if (Ai + ii < M && Ak + kk < K_V) {
    //         assert(A_shared[ii][kk].x == A[(Ai + ii) * K_V + Ak + kk].x);
    //         assert(A_shared[ii][kk].y == A[(Ai + ii) * K_V + Ak + kk].y);
    //         assert(A_shared[ii][kk].z == A[(Ai + ii) * K_V + Ak + kk].z);
    //         assert(A_shared[ii][kk].w == A[(Ai + ii) * K_V + Ak + kk].w);
    //       } else {
    //         assert(A_shared[ii][kk].x == 0.f);
    //         assert(A_shared[ii][kk].y == 0.f);
    //         assert(A_shared[ii][kk].z == 0.f);
    //         assert(A_shared[ii][kk].w == 0.f);
    //       }
    //     }
    //   }
    // }

    // validate B
    // if (threadIdx.x == 0 && threadIdx.y == 0) {
    //   int Bk = tk;
    //   for (int kk = 0; kk < BLOCK_TILE_K; ++kk) {
    //     int Bj = (BLOCK_TILE_N_V) * blockIdx.x;
    //     for (int jj = 0; jj < (BLOCK_TILE_N_V); ++jj) {
    //       if (Bk + kk < K && Bj + jj < N_V) {
    //         assert(B_shared[kk][jj].x == B[(Bk + kk) * N_V + Bj + jj].x);
    //         assert(B_shared[kk][jj].y == B[(Bk + kk) * N_V + Bj + jj].y);
    //         assert(B_shared[kk][jj].z == B[(Bk + kk) * N_V + Bj + jj].z);
    //         assert(B_shared[kk][jj].w == B[(Bk + kk) * N_V + Bj + jj].w);
    //       } else {
    //         assert(B_shared[kk][jj].x == 0.f);
    //         assert(B_shared[kk][jj].y == 0.f);
    //         assert(B_shared[kk][jj].z == 0.f);
    //         assert(B_shared[kk][jj].w == 0.f);
    //       }
    //     }
    //   }
    // }

    for (int k = 0; k < BLOCK_TILE_K_V; ++k) {

      // thread tile repeat row
      for (int ttr = 0; ttr < NUM_THREAD_TILES_PER_WARP_TILE_M; ++ttr) {
        // warp tile row + ttr stride + thread tile row
        const int ttri = WARP_Y * WARP_TILE_M \
                         + ttr * (WARP_TILE_M / NUM_THREAD_TILES_PER_WARP_TILE_M) \
                         + THREAD_Y_IN_WARP * THREAD_TILE_M;
        for (int ii = 0; ii < THREAD_TILE_M; ++ii) {
          a_reg[ttr][ii] = A_shared[ttri + ii][k];
        }
      }

      // thread tile repeat col
      for (int ttc = 0; ttc < NUM_THREAD_TILES_PER_WARP_TILE_N; ++ttc) {
        // warp tile col + ttc stride + thread tile col
        const int ttcj = WARP_X * WARP_TILE_N_V \
                         + ttc * (WARP_TILE_N_V / NUM_THREAD_TILES_PER_WARP_TILE_N) \
                         + THREAD_X_IN_WARP * THREAD_TILE_N_V;
        for (int jj = 0; jj < THREAD_TILE_N_V; ++jj) {
          b_reg[ttc][0][jj] = B_shared[V * k + 0][ttcj + jj];
          b_reg[ttc][1][jj] = B_shared[V * k + 1][ttcj + jj];
          b_reg[ttc][2][jj] = B_shared[V * k + 2][ttcj + jj];
          b_reg[ttc][3][jj] = B_shared[V * k + 3][ttcj + jj];
        }
      }

      // outer product
      for (int ttr = 0; ttr < NUM_THREAD_TILES_PER_WARP_TILE_M; ++ttr) {
        for (int ttc = 0; ttc < NUM_THREAD_TILES_PER_WARP_TILE_N; ++ttc) {
          for (int ii = 0; ii < THREAD_TILE_M; ++ii) {
            for (int jj = 0; jj < THREAD_TILE_N_V; ++jj) {
              c_reg[ttr][ttc][ii][jj].x += a_reg[ttr][ii].x * b_reg[ttc][0][jj].x;
              c_reg[ttr][ttc][ii][jj].y += a_reg[ttr][ii].x * b_reg[ttc][0][jj].y;
              c_reg[ttr][ttc][ii][jj].z += a_reg[ttr][ii].x * b_reg[ttc][0][jj].z;
              c_reg[ttr][ttc][ii][jj].w += a_reg[ttr][ii].x * b_reg[ttc][0][jj].w;

              c_reg[ttr][ttc][ii][jj].x += a_reg[ttr][ii].y * b_reg[ttc][1][jj].x;
              c_reg[ttr][ttc][ii][jj].y += a_reg[ttr][ii].y * b_reg[ttc][1][jj].y;
              c_reg[ttr][ttc][ii][jj].z += a_reg[ttr][ii].y * b_reg[ttc][1][jj].z;
              c_reg[ttr][ttc][ii][jj].w += a_reg[ttr][ii].y * b_reg[ttc][1][jj].w;

              c_reg[ttr][ttc][ii][jj].x += a_reg[ttr][ii].z * b_reg[ttc][2][jj].x;
              c_reg[ttr][ttc][ii][jj].y += a_reg[ttr][ii].z * b_reg[ttc][2][jj].y;
              c_reg[ttr][ttc][ii][jj].z += a_reg[ttr][ii].z * b_reg[ttc][2][jj].z;
              c_reg[ttr][ttc][ii][jj].w += a_reg[ttr][ii].z * b_reg[ttc][2][jj].w;

              c_reg[ttr][ttc][ii][jj].x += a_reg[ttr][ii].w * b_reg[ttc][3][jj].x;
              c_reg[ttr][ttc][ii][jj].y += a_reg[ttr][ii].w * b_reg[ttc][3][jj].y;
              c_reg[ttr][ttc][ii][jj].z += a_reg[ttr][ii].w * b_reg[ttc][3][jj].z;
              c_reg[ttr][ttc][ii][jj].w += a_reg[ttr][ii].w * b_reg[ttc][3][jj].w;
            }
          }
        }
      }

    }
    // sync after use
    __syncthreads();
  }

  // copy back
  for (int ttr = 0; ttr < NUM_THREAD_TILES_PER_WARP_TILE_M; ++ttr) {
    for (int ttc = 0; ttc < NUM_THREAD_TILES_PER_WARP_TILE_N; ++ttc) {
      for (int ii = 0; ii < THREAD_TILE_M; ++ii) {
        for (int jj = 0; jj < THREAD_TILE_N_V; ++jj) {
          const int i = blockIdx.y * BLOCK_TILE_M + WARP_Y * WARP_TILE_M + ttr * (WARP_TILE_M / NUM_THREAD_TILES_PER_WARP_TILE_M) + THREAD_Y_IN_WARP * THREAD_TILE_M + ii;
          const int j = blockIdx.x * BLOCK_TILE_N_V + WARP_X * WARP_TILE_N_V + ttc * (WARP_TILE_N_V / NUM_THREAD_TILES_PER_WARP_TILE_N) + THREAD_X_IN_WARP * THREAD_TILE_N_V + jj;
          C[i * N_V + j] = c_reg[ttr][ttc][ii][jj];
        }
      }
    }
  }
}

int main(int argc, char *argv[]) {
  assert(K % V == 0); assert(N % V == 0);

  A = (float *)malloc(M * K * sizeof(float));
  B = (float *)malloc(K * N * sizeof(float));
  C = (float *)malloc(M * N * sizeof(float));
  C_ans = (float *)malloc(M * N * sizeof(float));

  CHECK_CUDA(hipMalloc(&A_cuda, M * K * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_cuda, K * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_cuda, M * N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_cublas, M * N * sizeof(float)));

  for (int i = 0; i < M * K; ++i) {
    A[i] = 2 * (rand() / (double)RAND_MAX);
  }
  for (int i = 0; i < K * N; ++i) {
    B[i] = 2 * (rand() / (double)RAND_MAX);
  }

  CHECK_CUDA(hipMemcpy(A_cuda, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B_cuda, B, K * N * sizeof(float), hipMemcpyHostToDevice));

  printf("Running kernel\n");
#ifdef WARMUP
  {
    for (int ii = 0; ii < 10; ++ii) {
      dim3 blockDim{ BLOCK_N, BLOCK_M, 1 };
      dim3 gridDim{ (unsigned int)(N + BLOCK_TILE_N - 1) / BLOCK_TILE_N, (unsigned int)(M + BLOCK_TILE_M - 1) / BLOCK_TILE_M, 1 };
      mm32wv << < gridDim, blockDim >> > ((float4*)A_cuda, (float4*)B_cuda, (float4*)C_cuda, M, K, N);
      CHECK_CUDA(hipGetLastError());
    }
    CHECK_CUDA(hipDeviceSynchronize());
  }
#endif

  struct timespec s, e;
  clock_gettime(CLOCK_MONOTONIC, &s);
  {
    dim3 blockDim{ BLOCK_N, BLOCK_M, 1 };
    dim3 gridDim{ (unsigned int)(N + BLOCK_TILE_N - 1) / BLOCK_TILE_N, (unsigned int)(M + BLOCK_TILE_M - 1) / BLOCK_TILE_M, 1 };
    mm32wv << < gridDim, blockDim >> > ((float4*)A_cuda, (float4*)B_cuda, (float4*)C_cuda, M, K, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
  }

  clock_gettime(CLOCK_MONOTONIC, &e);
  double elapsed = (e.tv_sec - s.tv_sec) + ((double)e.tv_nsec - s.tv_nsec) / 1000000000.;
  double bw = 2.0 * M * K * N / 1000000000. / elapsed;
  printf("elapsed time: %lfs, bandwidth: %lf GB/s\n", elapsed, bw);
  CHECK_CUDA(hipMemcpy(C, C_cuda, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // cublas verify
  if (argc == 2) {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    struct timespec s, e;
    clock_gettime(CLOCK_MONOTONIC, &s);
    {
      printf("Running cublas\n");
      float alpha = 1.f;
      float beta = 0.f;
      CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B_cuda, N, A_cuda, K, &beta, C_cublas, N));
      CHECK_CUDA(hipDeviceSynchronize());
    }
    clock_gettime(CLOCK_MONOTONIC, &e);
    double elapsed = (e.tv_sec - s.tv_sec) + ((double)e.tv_nsec - s.tv_nsec) / 1000000000.;
    double cublas_bw = 2.0 * M * K * N / 1000000000. / elapsed;
    printf("elapsed time: %lfs, bandwidth: %lf GB/s\n", elapsed, cublas_bw);

    printf("Kernel / cuBlas = %lf / %lf = %lf %%\n", bw, cublas_bw, bw / cublas_bw * 100);
    CHECK_CUDA(hipMemcpy(C_ans, C_cublas, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    for (int i = 0; i < M; ++i) {
      for (int j = 0; j < N; ++j) {
        if (fabs((C[i * N + j] - C_ans[i * N + j]) / C[i * N + j]) >= EPS) {
          printf("Validation Failed! C[%d, %d]: %f %f\n", i, j, C[i * N + j], C_ans[i * N + j]);
          exit(1);
        }
      }
    }
    printf("Verification Success!\n");
  }
}